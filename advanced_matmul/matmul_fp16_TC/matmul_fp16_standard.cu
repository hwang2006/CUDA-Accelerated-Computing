#include "hip/hip_runtime.h"
// matmul_fp16_standard.cu
#include <cstdio>
#include <hip/hip_runtime.h>
#include "matmul.h" // Header declaring matmul_init, matmul, matmul_cleanup

// Error checking macro
#define CHECK_CUDA(call)    do {    hipError_t status_ = call;    if (status_ != hipSuccess) {      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString(status_));      exit(EXIT_FAILURE);    }  } while (0)

// Block size for standard CUDA kernel
#define BLOCK_SIZE 32

// Kernel: Standard Matrix Multiplication
__global__ void matmul_kernel_standard(half *A, half *B, float *C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) {
            sum += (float)(A[row * K + i] * B[i * N + col]);
        }
        C[row * N + col] = sum;
    }
}

// Static device pointers for A, B, and C
static half *A_gpu_standard, *B_gpu_standard;
static float *C_gpu_standard;

// Host function: Copy matrices to GPU and launch the standard kernel
void matmul_standard(half *_A, half *_B, float *_C, int M, int N, int K) {
    CHECK_CUDA(hipMemcpy(A_gpu_standard, _A, M * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(B_gpu_standard, _B, K * N * sizeof(half), hipMemcpyHostToDevice));

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matmul_kernel_standard<<<gridDim, blockDim>>>(A_gpu_standard, B_gpu_standard, _C, M, N, K);

    CHECK_CUDA(hipGetLastError()); // Check kernel launch success
    CHECK_CUDA(hipMemcpy(_C, C_gpu_standard, M * N * sizeof(float), hipMemcpyDeviceToHost));
}

// Host function: Allocate memory on device for standard kernel
void matmul_init_standard(int M, int N, int K) {
    CHECK_CUDA(hipMalloc(&A_gpu_standard, M * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&B_gpu_standard, K * N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&C_gpu_standard, M * N * sizeof(float)));
}

// Host function: Free memory on device for standard kernel
void matmul_cleanup_standard(half *_A, half *_B, float *_C, int M, int N, int K) {
    CHECK_CUDA(hipFree(A_gpu_standard));
    CHECK_CUDA(hipFree(B_gpu_standard));
    CHECK_CUDA(hipFree(C_gpu_standard));
}
