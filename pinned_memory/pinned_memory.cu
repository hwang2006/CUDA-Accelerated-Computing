#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

float measure_bandwidth_async(const char* label, void* dst, const void* src,
                size_t bytes, hipMemcpyKind kind, int repetitions, hipStream_t stream) {
  hipEvent_t start, stop;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&stop));

  // Warm-up
  CHECK_CUDA(hipMemcpyAsync(dst, src, bytes, kind, stream));
  CHECK_CUDA(hipStreamSynchronize(stream));

  CHECK_CUDA(hipEventRecord(start, stream));
  for (int i = 0; i < repetitions; ++i) {
    CHECK_CUDA(hipMemcpyAsync(dst, src, bytes, kind, stream));
  }
  CHECK_CUDA(hipEventRecord(stop, stream));
  CHECK_CUDA(hipEventSynchronize(stop));

  float ms = 0.0f;
  CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));

  float avg_time = ms / repetitions;
  float bandwidth = (bytes / avg_time / 1e6);  // GB/s

  printf("%-30s: %7.2f GB/s (%zu bytes, %d reps)\n", label, bandwidth, bytes, repetitions);

  CHECK_CUDA(hipEventDestroy(start));
  CHECK_CUDA(hipEventDestroy(stop));
  return bandwidth;
}

int main() {
  std::vector<size_t> sizes = {1 << 10, 1 << 12, 1 << 14, 1 << 16, 1<< 18, 1 << 20, 1 << 22, 1 << 24, 1 << 26, 1 << 28};// 1MB, 4MB, 16MB, 64MB, 256MB
  const int repetitions = 10;

  hipStream_t stream;
  CHECK_CUDA(hipStreamCreate(&stream));

  for (size_t bytes : sizes) {
    printf("\n===== Buffer size: %zu bytes (%.2f MB) =====\n", bytes, bytes / (1024.0 * 1024.0));

    // Allocate device memory
    int *d_mem;
    CHECK_CUDA(hipMalloc(&d_mem, bytes));

    // 1. Pageable memory
    {
      int *h_pageable = (int*) malloc(bytes);
      if (!h_pageable) {
        fprintf(stderr, "Failed to allocate pageable memory\n");
        return EXIT_FAILURE;
      }

      measure_bandwidth_async("H2D Pageable (async)", d_mem, h_pageable, bytes, hipMemcpyHostToDevice, repetitions, stream);
      measure_bandwidth_async("D2H Pageable (async)", h_pageable, d_mem, bytes, hipMemcpyDeviceToHost, repetitions, stream);

      free(h_pageable);
    }

    // 2. Pinned memory
    {
      int *h_pinned;
      CHECK_CUDA(hipHostMalloc(&h_pinned, bytes, hipHostMallocDefault));

      measure_bandwidth_async("H2D Pinned (async)", d_mem, h_pinned, bytes, hipMemcpyHostToDevice, repetitions, stream);
      measure_bandwidth_async("D2H Pinned (async)", h_pinned, d_mem, bytes, hipMemcpyDeviceToHost, repetitions, stream);

      CHECK_CUDA(hipHostFree(h_pinned));
    }

    CHECK_CUDA(hipFree(d_mem));
  }

  CHECK_CUDA(hipStreamDestroy(stream));
  return 0;
}
