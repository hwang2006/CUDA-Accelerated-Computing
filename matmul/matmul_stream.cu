#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

#define LOOP_I(_loop) for(int i=0; i < _loop; i++)

#define COL_SIZE (32)
#define ROW_SIZE (32)

//#define COL_SIZE (64)
//#define ROW_SIZE (16)

//#define BLOCKS 8
#define BLOCKS 16

static int Mbegin[BLOCKS], Mend[BLOCKS];
static hipStream_t upload_stream, download_stream, calc_stream;
static hipEvent_t upload_events[BLOCKS], calc_events[BLOCKS];

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void naive_cpu_matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[i * N + j] += _A[i * K + k] * _B[k * N + j];
      }
    }
  }
}
/*
__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K) {

   int ix = threadIdx.x + blockIdx.x * blockDim.x; //column
   int iy = threadIdx.y + blockIdx.y * blockDim.y; //row
 
   if (iy >= M || ix >= N) return;
   float sum = 0.0f;
   for (int k = 0; k < K; k++) sum += A[iy * K + k] * B[k * N + ix];
   C[iy * N + ix] = sum;
}
*/
/*
// matmul_kernel_xRow
__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                                     int K) {
  int i = blockDim.x * blockIdx.x + threadIdx.x; //col
  int j = blockDim.y * blockIdx.y + threadIdx.y; //row
  if (i >= N || j >= M) return;
  float sum = 0.0;
  for (int k = 0; k < K; ++k) sum += A[j * K + k] * B[k * N + i];
  C[j * N + i] = sum;
}
*/

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K) {
    // Calculate the row and column for this thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within valid range
    if (col >= N || row >= M) return;

    float sum = 0.0f;
    for (int k = 0; k < K; k++) sum += A[row * K + k] * B[k * N + col];
    C[row * N + col] = sum;

}


__global__ void matmul_kernel_xRow(float *A, float *B, float *C, int M, int N, int K) {

   int row = threadIdx.x + blockIdx.x * blockDim.x; 
   int col = threadIdx.y + blockIdx.y * blockDim.y; 

   //if ( col >= M || row >= N) return;
   if ( col >= N || row >= M) return;
   float sum = 0.0f;
   for (int k = 0; k < K; k++) sum += A[row * K + k] * B[k * N + col];
   C[row * N + col] = sum;
}


/*
__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K) {
    // Calculate the row and column for this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within valid range
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}
*/

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // Remove this line after you complete the matmul on GPU
  //naive_cpu_matmul(_A, _B, _C, M, N, K);

  // (TODO) Upload A and B matrix to GPU
  //CHECK_CUDA(hipMemcpy(A_gpu, _A, M * K * sizeof(float), hipMemcpyHostToDevice));
  //CHECK_CUDA(hipMemcpy(B_gpu, _B, K * N * sizeof(float), hipMemcpyHostToDevice));  

  // upload stream
  CHECK_CUDA(hipMemcpyAsync(B_gpu, _B, K * N * sizeof(float),
                             hipMemcpyHostToDevice, upload_stream));
  LOOP_I(BLOCKS)
  { 
    CHECK_CUDA(hipMemcpyAsync(&A_gpu[Mbegin[i] * K], &_A[Mbegin[i] * K],
                               (Mend[i] - Mbegin[i]) * K * sizeof(float),
                               hipMemcpyHostToDevice, upload_stream));
    CHECK_CUDA(hipEventRecord(upload_events[i], upload_stream));
  }

  // calc stream
  LOOP_I(BLOCKS)
  {
    //dim3 blockDim(16, 16);
    //dim3 gridDim((N + 16 - 1) / 16, (Mend[i] - Mbegin[i] + 16 - 1) / 16);


    dim3 blockDim(COL_SIZE, ROW_SIZE);
    //dim3 gridDim((N + COL_SIZE - 1)/COL_SIZE, (M + ROW_SIZE -1)/ROW_SIZE);
    dim3 gridDim((N + blockDim.x - 1)/blockDim.x, (M + blockDim.y -1)/blockDim.y);

    CHECK_CUDA(hipStreamWaitEvent(calc_stream, upload_events[i]));
    matmul_kernel<<<gridDim, blockDim, 0, calc_stream>>>(
        &A_gpu[Mbegin[i] * K], B_gpu, &C_gpu[Mbegin[i] * N],
        (Mend[i] - Mbegin[i]), N, K);
    CHECK_CUDA(hipEventRecord(calc_events[i], calc_stream));

  }

  // download stream
  LOOP_I(BLOCKS)
  {
    CHECK_CUDA(hipStreamWaitEvent(download_stream, calc_events[i]));
    CHECK_CUDA(hipMemcpyAsync(&_C[Mbegin[i] * N], &C_gpu[Mbegin[i] * N],
                               (Mend[i] - Mbegin[i]) * N * sizeof(float),
                               hipMemcpyDeviceToHost, download_stream)); 
  }
  // (TODO) Launch kernel on a GPU
  //dim3 blockDim(COL_SIZE, ROW_SIZE);
  //dim3 gridDim((N + COL_SIZE - 1)/COL_SIZE, (M + ROW_SIZE -1)/ROW_SIZE);
  //dim3 gridDim((N + blockDim.x - 1)/blockDim.x, (M + blockDim.y -1)/blockDim.y);

  //dim3 blockDim(32, 32);
  //dim3 gridDim((M + 32 - 1) / 32, (N + 32 - 1) / 32);
  //fprintf(stdout, "Grid dimensions: %d x %d blocks of %d x %d threads\n",
  //         			gridDim.x, gridDim.y, blockDim.x, blockDim.y);
  //fflush(stdout);
  //matmul_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);


  //(TODO) Launch kernel with xRow on a GPU
  //dim3 gridDim((M + COL_SIZE - 1)/COL_SIZE, (N + ROW_SIZE -1)/ROW_SIZE);
  //dim3 gridDim((M + blockDim.x - 1)/blockDim.x, (N + blockDim.y -1)/blockDim.y);
  //matmul_kernel_xRow<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);
  //CHECK_CUDA(hipGetLastError());  

  // (TODO) Download C matrix from GPU
  //CHECK_CUDA(hipMemcpy(_C, C_gpu, M * N * sizeof(float), hipMemcpyDeviceToHost));  

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));
  
  LOOP_I(BLOCKS)
  {
    Mbegin[i] = M / BLOCKS * i;
    Mend[i] = M / BLOCKS * (i + 1);
    if (i == BLOCKS - 1) Mend[i] = M;
  }

  //Create streams
  CHECK_CUDA(hipStreamCreate(&upload_stream));
  CHECK_CUDA(hipStreamCreate(&download_stream));
  CHECK_CUDA(hipStreamCreate(&calc_stream));

  //Create events
  LOOP_I(BLOCKS)
  {
   CHECK_CUDA(hipEventCreate(&upload_events[i]));
   CHECK_CUDA(hipEventCreate(&calc_events[i]));
  }
  
  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));
  CHECK_CUDA(hipStreamDestroy(upload_stream));
  CHECK_CUDA(hipStreamDestroy(download_stream));
  CHECK_CUDA(hipStreamDestroy(calc_stream));
  LOOP_I(BLOCKS)
  {
    CHECK_CUDA(hipEventDestroy(upload_events[i]));
    CHECK_CUDA(hipEventDestroy(calc_events[i]));
  }

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
