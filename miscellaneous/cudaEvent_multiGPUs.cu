#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <DS_timer.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define LOOP_I(_loop) for(int i=0; i < _loop; i++)

//#define NUM_BLOCK (128*1024)
#define NUM_BLOCK (16 * 1024)
#define NUM_T_IN_B 1024
#define ARRAY_SIZE (NUM_T_IN_B*NUM_BLOCK)

#define NUM_STREAMS 4
//#define NUM_STREAMS 1

__global__ void myKernel2(int *_in, int *_out)
{
    int tID = blockDim.x * blockIdx.x + threadIdx.x;

    int temp = 0;
    for (int i = 0; i < 250; i++) {
        temp = (temp + _in[tID] * 5) % 10;
    }
    _out[tID] = temp;
}

int main(void)
{
    DS_timer timer(2);
    timer.setTimerName(0, "CPU code");
    timer.setTimerName(1, "GPU kernel");

    int *in = NULL, *out = NULL, *out2 = NULL;

    hipHostMalloc(&in, sizeof(int)*ARRAY_SIZE);
    memset(in, 0, sizeof(int)*ARRAY_SIZE);

    hipHostMalloc(&out, sizeof(int)*ARRAY_SIZE);
    memset(out, 1, sizeof(int)*ARRAY_SIZE);

    hipHostMalloc(&out2, sizeof(int)*ARRAY_SIZE);
    memset(out2, 0, sizeof(int)*ARRAY_SIZE);

    LOOP_I(ARRAY_SIZE)
        in[i] = rand() % 10;

    // CPU-side equivalent computation for validation
    printf("CPU-side equivalent computation for validation...\n"); fflush(stdout);

    timer.onTimer(0);
#pragma omp parallel for num_threads(32)
    LOOP_I(ARRAY_SIZE) {
        int temp = 0;
        for (int j = 0; j < 250; j++) {
            temp = (temp + in[i] * 5) % 10;
        }
        out[i] = temp;
    }
    timer.offTimer(0);
    printf("Done!!\n");

    hipStream_t stream[NUM_STREAMS];
    hipEvent_t start[NUM_STREAMS], end[NUM_STREAMS];

    int* dIn[NUM_STREAMS];
    int* dOut[NUM_STREAMS];

    int ngpus;
    hipGetDeviceCount(&ngpus);

    int chunkSize = ARRAY_SIZE / NUM_STREAMS;

    LOOP_I(NUM_STREAMS) {
        int devId = i % ngpus;
        hipSetDevice(devId);

        hipStreamCreate(&stream[i]);
        hipEventCreate(&start[i]);
        hipEventCreate(&end[i]);

        hipMalloc(&dIn[i], sizeof(int) * chunkSize);
        hipMalloc(&dOut[i], sizeof(int) * chunkSize);
    }

    timer.onTimer(1);

#pragma omp parallel for num_threads(NUM_STREAMS)
    LOOP_I(NUM_STREAMS)
    {
        int devId = i % ngpus;
        hipSetDevice(devId);

        int offset = chunkSize * i;

        printf("Launching Stream[%d] on Device[%d]\n", i, devId);

        hipEventRecord(start[i], stream[i]);

        hipMemcpyAsync(dIn[i], in + offset, sizeof(int)*chunkSize, hipMemcpyHostToDevice, stream[i]);
        myKernel2<<<NUM_BLOCK / NUM_STREAMS, NUM_T_IN_B, 0, stream[i]>>>(dIn[i], dOut[i]);
        hipMemcpyAsync(out2 + offset, dOut[i], sizeof(int)*chunkSize, hipMemcpyDeviceToHost, stream[i]);

        hipEventRecord(end[i], stream[i]);

        //hipStreamSynchronize(stream[i]); // Optional for accurate per-stream timing
        //hipEventSynchronize(end[i]);
    }

    // Wait for all GPUs to finish
    LOOP_I(NUM_STREAMS) {
        hipSetDevice(i % ngpus);
        hipDeviceSynchronize();
    }

    timer.offTimer(1);
    timer.printTimer();

    float totalStreamTime = 0;
    LOOP_I(NUM_STREAMS) {
        float time = 0;
        hipEventSynchronize(end[i]);
        hipEventElapsedTime(&time, start[i], end[i]);
        totalStreamTime += time;
        printf("Stream[%d] : %f ms\n", i, time);
    }
    //printf("Sum of all stream GPU times = %f ms\n", totalStreamTime);

    bool mismatch_found = false;

    LOOP_I(ARRAY_SIZE) {
        if (out[i] != out2[i]) {
           printf("Data mismatch at index %d: expected %d, got %d\n", i, out[i], out2[i]);
           mismatch_found = true;
           break;
        }
    }

    if (!mismatch_found) {
        printf("Validation PASSED: All results match!\n");
    } else {
        printf("Validation FAILED: At least one mismatch found.\n");
    }


    LOOP_I(NUM_STREAMS) {
        hipSetDevice(i % ngpus);
        hipFree(dIn[i]);
        hipFree(dOut[i]);

        hipStreamDestroy(stream[i]);
        hipEventDestroy(start[i]);
        hipEventDestroy(end[i]);
    }

    hipHostFree(in);
    hipHostFree(out);
    hipHostFree(out2);

    return 0;
}
