/**
This is an exmple code used in the CUDA Lecture 5 (Quick Lab. 12-1) <br>
@author : Duksu Kim
*/

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

int main(void) {
	int ngpus;
	hipGetDeviceCount(&ngpus);

	for (int i = 0; i < ngpus; i++) {
		hipDeviceProp_t devProp;

		hipGetDeviceProperties(&devProp, i);
		printf("Device[%d](%s) compute capability : %d.%d.\n"
			, i, devProp.name, devProp.major, devProp.minor);
	}
}
