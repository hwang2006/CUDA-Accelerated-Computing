#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define THREADS_PER_BLOCK 1024
#define NUM_BLOCK (1024 * 1024)
#define NUM_DATA (THREADS_PER_BLOCK * NUM_BLOCK)
#define NUM_STREAMS_PER_GPU 4

__global__ void vecAdd(int *_a, int *_b, int *_c, int _size) {
    //int tID = blockIdx.x * blockDim.x + threadIdx.x;
    size_t tID = blockIdx.x * blockDim.x + threadIdx.x;
    if (tID < _size)
        _c[tID] = _a[tID] + _b[tID];
}

int main(void)
{
    DS_timer timer(2);
    timer.setTimerName(0, "CUDA Total");
    timer.setTimerName(1, "VectorSum on Host");

    int numDevices;
    hipGetDeviceCount(&numDevices);
    if (numDevices == 0) {
        printf("No CUDA-capable GPU found.\n");
        return -1;
    }

    size_t memSize = sizeof(int) * NUM_DATA;
    printf("%zu elements, memSize = %zu bytes, using %d GPUs\n", NUM_DATA, memSize, numDevices);

    // Host allocations (pinned)
    int *a, *b, *c, *h_c;
    hipHostMalloc(&a, memSize);
    hipHostMalloc(&b, memSize);
    hipHostMalloc(&c, memSize);
    h_c = (int *)malloc(memSize);

    //for (int i = 0; i < NUM_DATA; i++) {
    for (size_t i = 0; i < NUM_DATA; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // CPU reference
    timer.onTimer(1);
    //for (int i = 0; i < NUM_DATA; i++)
    for (size_t i = 0; i < NUM_DATA; i++)
        h_c[i] = a[i] + b[i];
    timer.offTimer(1);

    // Divide work across GPUs
    size_t chunkPerGPU = NUM_DATA / numDevices;
    size_t chunkPerStream = chunkPerGPU / NUM_STREAMS_PER_GPU;
    size_t chunkBytes = chunkPerStream * sizeof(int);
    dim3 dimBlock(THREADS_PER_BLOCK);
    dim3 dimGrid((chunkPerStream + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    timer.onTimer(0);

    for (int dev = 0; dev < numDevices; dev++) {
        hipSetDevice(dev);

        // Allocate per-GPU memory
        int *d_a, *d_b, *d_c;
        hipMalloc(&d_a, chunkPerGPU * sizeof(int));
        hipMalloc(&d_b, chunkPerGPU * sizeof(int));
        hipMalloc(&d_c, chunkPerGPU * sizeof(int));

        hipStream_t streams[NUM_STREAMS_PER_GPU];
        hipEvent_t start[NUM_STREAMS_PER_GPU], end[NUM_STREAMS_PER_GPU];

        for (int s = 0; s < NUM_STREAMS_PER_GPU; s++) {
            hipStreamCreate(&streams[s]);
            hipEventCreate(&start[s]);
            hipEventCreate(&end[s]);

            //int globalOffset = dev * chunkPerGPU + s * chunkPerStream;
            size_t globalOffset = dev * chunkPerGPU + s * chunkPerStream;

            hipEventRecord(start[s], streams[s]);
            hipMemcpyAsync(d_a + s * chunkPerStream, a + globalOffset, chunkBytes, hipMemcpyHostToDevice, streams[s]);
            hipMemcpyAsync(d_b + s * chunkPerStream, b + globalOffset, chunkBytes, hipMemcpyHostToDevice, streams[s]);

            vecAdd<<<dimGrid, dimBlock, 0, streams[s]>>>(
                d_a + s * chunkPerStream,
                d_b + s * chunkPerStream,
                d_c + s * chunkPerStream,
                chunkPerStream
            );

            hipMemcpyAsync(c + globalOffset, d_c + s * chunkPerStream, chunkBytes, hipMemcpyDeviceToHost, streams[s]);
            hipEventRecord(end[s], streams[s]);
        }

        for (int s = 0; s < NUM_STREAMS_PER_GPU; s++) {
            hipStreamSynchronize(streams[s]);
            float time = 0;
            hipEventElapsedTime(&time, start[s], end[s]);
            printf("Device[%d] Stream[%d]: %.2f ms\n", dev, s, time);
            hipEventDestroy(start[s]);
            hipEventDestroy(end[s]);
            hipStreamDestroy(streams[s]);
        }

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    timer.offTimer(0);
    timer.printTimer();

    // Verify
    bool correct = true;
    //for (int i = 0; i < NUM_DATA; i++) {
    for (size_t i = 0; i < NUM_DATA; i++) {
        if (h_c[i] != c[i]) {
            //printf("Mismatch at %d: CPU = %d, GPU = %d\n", i, h_c[i], c[i]);
            printf("Mismatch at %zu: CPU = %d, GPU = %d\n", i, h_c[i], c[i]);
            correct = false;
            break;
        }
    }
    if (correct)
        printf("GPU works well across %d devices!\n", numDevices);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    free(h_c);

    return 0;
}
